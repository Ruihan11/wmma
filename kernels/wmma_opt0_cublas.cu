#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

// cuBLAS wrapper kernel for performance comparison
class CuBLASWrapper {
private:
    static hipblasHandle_t handle;
    static bool initialized;

public:
    static void initialize() {
        if (!initialized) {
            hipblasCreate(&handle);
            hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
            initialized = true;
        }
    }
    
    static void cleanup() {
        if (initialized) {
            hipblasDestroy(handle);
            initialized = false;
        }
    }
    
    static void gemm(half* d_a, half* d_b, float* d_c, int m, int n, int k) {
        const float alpha = 1.0f, beta = 0.0f;
        
        hipblasGemmEx(handle, 
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, m, k,
                    &alpha,
                    d_b, HIP_R_16F, n,
                    d_a, HIP_R_16F, k,
                    &beta,
                    d_c, HIP_R_32F, n,
                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
    
    static void warmup(half* d_a, half* d_b, float* d_c, int m, int n, int k) {
        // Perform warmup runs
        for (int i = 0; i < 3; i++) {
            gemm(d_a, d_b, d_c, m, n, k);
        }
        hipDeviceSynchronize();
    }
};

// Static member initialization
hipblasHandle_t CuBLASWrapper::handle;
bool CuBLASWrapper::initialized = false;

extern "C" {
    void launch_wmma_opt0_cublas_init() {
        CuBLASWrapper::initialize();
    }
    
    void launch_wmma_opt0_cublas_cleanup() {
        CuBLASWrapper::cleanup();
    }
    
    void launch_wmma_opt0_cublas_warmup(half* d_a, half* d_b, float* d_c, int m, int n, int k) {
        CuBLASWrapper::warmup(d_a, d_b, d_c, m, n, k);
    }
    
    void launch_wmma_opt0_cublas(half* d_a, half* d_b, float* d_c, int m, int n, int k, hipStream_t stream = 0) {
        CuBLASWrapper::gemm(d_a, d_b, d_c, m, n, k);
    }
}